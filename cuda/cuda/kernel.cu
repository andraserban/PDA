#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

__constant__ float arraySum[255];
__global__ void sumOfThePixels(unsigned char *source, int width, int height,unsigned char *destination)
{
	// Calculate our pixel's location
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;

	float sum = 0.0f;
	float tmp_sum[255];//alocat

	for (int i = 0; i <= width; i++)
	{
		tmp_sum[x] += source[x][i] ;
	}
	for (int j = 0; j <= ; j++)
		//parcurg tmp_sum => suma finala
		sum += tmp_sum[j];

	destination[(y * width) + x] = (unsigned char)sum;
}
int main(int argc, char** argv)
{
	
	cv::VideoCapture camera(0);
	cv::Mat          frame;
	if (!camera.isOpened())
		return -1;
}